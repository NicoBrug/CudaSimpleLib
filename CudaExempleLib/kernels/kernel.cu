#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "stdio.h"

#define N 10

__global__ void kernel_add(int* c, int* a, int* b, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

// Helper function for using CUDA to add vectors in parallel.
void add(int* c, const int* a, const int* b, int size) {
    int* dev_a = nullptr;
    int* dev_b = nullptr;
    int* dev_output = nullptr;

    // Allocate GPU buffers for three vectors (two input, one output)
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
    hipMalloc((void**)&dev_output, size * sizeof(int));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    kernel_add << <2, (size + 1) / 2 >> > (dev_output, dev_a, dev_b, size);

    hipDeviceSynchronize();

    // Copy output vector from GPU buffers to host memory
    hipMemcpy(c, dev_output, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_output);
}